#include "hip/hip_runtime.h"
// Parallel Implementation of LZSS with no wrapping
// This implemenetation is based on the approach given 
// given in sequential file written by Alfonso Laffont
// ########################################################################################

// Current Task: 
// In the search phase, we will use CUDA threads to search for matching substrings in parallel
// Will also launch a kernel In the encode phase.

// Why not CUDA streams?
// Using CUDA streams to overlap the data transfers with the computation
// CUDA streams allow you to execute multiple kernels concurrently on the GPU
// and overlap data transfers with the computation

// If we look at runtime, reading from disk is really quick and compression is really slow.
// So. It turns out that we aren't being bottlenecked by I/O and should thus not use streams

// Instead, focus on what is limitting compression > Algorithmic approach might not be ideal!


#include <cmath>
#include <iostream>
#include <sstream>
#include <string>
#include <fstream>
#include <vector>
#include <bitset>
//measure how long code takes
#include <chrono>
using namespace std;

// GLOBAL VARIABLES
#define MATCH_LENGTH_MASK (15)
#define WINDOW_SIZE (4095)

// ascii to char
void print_char(int v){
    std::cout << (char)v;
}
// get match distance using bit operations
uint16_t get_distance(uint16_t input){
    return (input >> 4); 
}

// get match length using bit operations
__host__ __device__ uint16_t get_length(uint16_t input) {
    return (input & 0xF);
}

void print_string(std::vector<unsigned char> s1, int start, int end){
    for(int i = start; i < end; i++){
        std::cout << s1.at(i);
    }
    std::cout << std::endl;
}

bool are_vectors_equal(std::vector<unsigned char> s1, std::vector<unsigned char> s2){
    if(s1.size() != s2.size()){
        printf("Strings aren't equal length\n");
        return 0;
    }

    for(int i = 0; i < s1.size(); i++){
        if(s1.at(i) != s2.at(i)){
            printf("string values mismatch\n");
            return 0;
        }
    }
    printf("Strings are equal!\n");
    return 1;
}

__global__ void find_longest_match_kernel(unsigned char *data, int i, uint16_t *longest_match_array, int len) {
    
    __shared__ bool terminate_early = false;

    uint orig_i = i;
    uint cur = threadIdx.x + max(0, i - WINDOW_SIZE);
    uint16_t orig_cur = cur;
    // create the struct for storing the longest word we are currently look at
    uint length_of_match = 0;
    uint longest_match = 0;
    uint start = 0;
    // printf("thread idx: %d, cur: %d i: %d\n", threadIdx.x, cur, i);
    while(cur < i && i < len && length_of_match < MATCH_LENGTH_MASK){
        // we have a match
        // printf("cur data: %c\n", data[cur]);
        // printf("i data: %c\n", data[i]);
        if(data[cur] == data[i]){
            // printf("equal!\n");
            cur++;
            i++;
            length_of_match++;
        }else{
            // we have a new longest match
            if (longest_match < length_of_match){
                // printf("new longest match!\n");
                longest_match = length_of_match;
                length_of_match = 0;
                start = cur;
            }
            break;
        }
    }
    if (longest_match <= 2){
        // printf("return 0\n");
        longest_match_array[orig_cur] = 0;
    }else{
        // printf("return not 0\n");
        uint16_t ret = 0;
        ret = ((orig_i - ((start - longest_match))) << 4) | (longest_match);
        longest_match_array[orig_cur] = ret;
    }
}

__global__ void Max_Interleaved_Addressing_Global(uint16_t* data, int data_size){
    int idx = threadIdx.x;
    if (idx < data_size){
        for(int stride = 1; stride < data_size; stride *= 2) {
            if (idx % (2*stride) == 0) {
                uint16_t lhs = get_length(data[idx]);
                uint16_t rhs = get_length(data[idx + stride]);
                data[idx] = lhs < rhs ? data[idx + stride] : data[idx];
            }
            __syncthreads();
        }
    }
}


// keep an array of longest matches. 
// have each index select the length of the match  
uint16_t find_longest_match(unsigned char *d_data, int input_size, int i){
    int number_of_threads = WINDOW_SIZE + 1;

    uint16_t *device_longest_match_array;
    uint longest_match_array_length = min(WINDOW_SIZE, i + 1);
    // set things to 0 if not useful so no need to worry
    hipMalloc(&device_longest_match_array, 2 * longest_match_array_length);
    // invoke our kernel
    find_longest_match_kernel<<<1, min(i + 1, number_of_threads)>>>(d_data, i, device_longest_match_array, input_size);
    // find max in using interleave
    Max_Interleaved_Addressing_Global<<<1, min(i + 1, number_of_threads)>>>(device_longest_match_array, input_size);
    // alloc host mem
    uint16_t *host_longest_match_array = (uint16_t*)malloc(2);
    // only need to copy one item
    hipMemcpy(host_longest_match_array, device_longest_match_array, 2, hipMemcpyDeviceToHost);

    hipFree(device_longest_match_array);

    return host_longest_match_array[0];
}


// unsigned char is the representation of bytes that we will be using
std::vector<unsigned char> compress(unsigned char *d_data, std::vector<unsigned char> data, std::vector<bool> &compress_flags, int input_size){
    int i = 0;
    std::vector<unsigned char> output_buffer;
    // loop through all char in text
    while (i < input_size){
        uint16_t longest_match = find_longest_match(d_data, input_size, i);
        uint16_t match_length = get_length(longest_match); 
        // item needs to be longer than 2 in order to save data
        if(match_length > 2){
            // 12 bytes = distance and 4 bytes is length
            uint16_t match_distance = get_distance(longest_match);
            uint16_t match_length = get_length(longest_match); 
            // indicate that we are compressing
            compress_flags[i] = 1;
            output_buffer.push_back((longest_match >> 8) & 0xFF);
            output_buffer.push_back(longest_match & 0xFF);

            i += match_length;
        }else{
            output_buffer.push_back(data.at(i));
            i += 1;
        }
    }
    return output_buffer;
}

std::vector<unsigned char> decompress(std::vector<unsigned char> data, std::vector<bool> &compress_flags){
    int compress_index = 0;
    uint data_index = 0;
    std::vector<unsigned char> output;

    while(data_index < data.size()){
        // item is compressed
        // printf("%d\n", compress_flags.at(compress_index));
        if(compress_flags.at(compress_index) == 1){
            uint16_t p1 = data[data_index];
            uint16_t p2 = data[data_index+1];
            uint16_t compressed_data = ((p1 << 8) & 0xFF00) | p2;
            uint16_t match_distance = get_distance(compressed_data);
            uint16_t match_length = get_length(compressed_data); 

            for(int i = 0; i < match_length; i++){
                output.push_back(output[((compress_index - match_distance) + i)]);
            }
            // two bytes for compressed
            data_index += 2;
            compress_index += match_length;
        }
        // item isn't compressed
        else{
            output.push_back(data[data_index]);
            // one byte for non compressed
            data_index += 1;
            compress_index += 1;
        }
    }
    return output;
}

__global__ void print_kernel() {
    printf("Hello from block %d thread idx: %d\n", blockIdx.x,threadIdx.x);
}


int main(int argc, char *argv[]) {
    // printf("%d",argc);
    // printf("%s", argv[1]);
    std::ifstream inFile;
    //open the input file
    inFile.open(argv[1]); 
    
    std::stringstream strStream;
    auto read_file_start_time = std::chrono::high_resolution_clock::now();
    //read the file
    strStream << inFile.rdbuf(); 
    //str holds the content of the file
    std::string s = strStream.str(); 
    // text input
    vector<unsigned char> input(s.begin(), s.end());

    // copy data into GPU
    uint input_size = ((int)input.size());
    unsigned char *d_data;
    hipMalloc(&d_data, input_size);
    hipMemcpy(d_data, input.data(), input_size, hipMemcpyHostToDevice);

    auto read_file_end_time = std::chrono::high_resolution_clock::now();


    // compression flags all init at 0
    vector<bool> compress_flags(input.size(), 0);
    
    auto compression_start_time = std::chrono::high_resolution_clock::now();
    auto compressed_buf = compress(d_data, input, compress_flags, input_size);
    auto compression_end_time = std::chrono::high_resolution_clock::now();
    hipFree(d_data);

    auto decompression_start_time = std::chrono::high_resolution_clock::now();
    auto output = decompress(compressed_buf, compress_flags);
    auto decompression_end_time = std::chrono::high_resolution_clock::now();


    bool res = are_vectors_equal(output, input);

    auto compression_duration = (std::chrono::duration_cast<std::chrono::microseconds>(compression_end_time - compression_start_time).count()/pow(10,6));
    auto decompression_duration = (std::chrono::duration_cast<std::chrono::microseconds>(decompression_end_time - decompression_start_time).count()/pow(10,6));
    auto read_duration = (std::chrono::duration_cast<std::chrono::microseconds>(read_file_end_time - read_file_start_time).count()/pow(10,6));
    // write to output file

    ofstream myfile;
    myfile.open("output.txt");

    myfile << "Reading Input & Copying to GPU took: " << read_duration << " seconds" << std::endl;
    myfile << "Compression took: " << compression_duration << " seconds" << std::endl;
    myfile << "Decompression took: " << decompression_duration << " seconds" << std::endl;
    myfile.close();

    return 1;
}
