// Parallel Implementation of LZSS with no wrapping
// This implemenetation is based on the approach given 
// given in sequential file written by Alfonso Laffont
// ########################################################################################

// Current Task: 
// In the search phase, we will use CUDA threads to search for matching substrings in parallel
// Will also launch a kernel In the encode phase.

// Why not CUDA streams?
// Using CUDA streams to overlap the data transfers with the computation
// CUDA streams allow you to execute multiple kernels concurrently on the GPU
// and overlap data transfers with the computation

// If we look at runtime, reading from disk is really quick and compression is really slow.
// So. It turns out that we aren't being bottlenecked by I/O and should thus not use streams

// Instead, focus on what is limitting compression > Algorithmic approach might not be ideal!



#include <hip/hip_runtime.h>
#include <cmath>
#include <iostream>
#include <sstream>
#include <string>
#include <fstream>
#include <vector>
#include <bitset>
//measure how long code takes
#include <chrono>
using namespace std;

// GLOBAL VARIABLES
#define MATCH_LENGTH_MASK (15)
#define WINDOW_SIZE (4095)
#define GRANULARITY (WINDOW_SIZE+1)

// ascii to char
void print_char(int v){
    std::cout << (char)v;
}
// get match distance using bit operations
uint16_t get_distance(uint16_t input){
    return (input >> 4); 
}

// get match length using bit operations
__host__ __device__ uint16_t get_length(uint16_t input) {
    return (input & 0xF);
}

void print_string(std::vector<unsigned char> s1, int start, int end){
    for(int i = start; i < end; i++){
        std::cout << s1.at(i);
    }
    std::cout << std::endl;
}

bool are_vectors_equal(std::vector<unsigned char> s1, std::vector<unsigned char> s2){
    if(s1.size() != s2.size()){
        printf("Strings aren't equal length\n");
        return 0;
    }

    for(int i = 0; i < s1.size(); i++){
        if(s1.at(i) != s2.at(i)){
            printf("string values mismatch\n");
            return 0;
        }
    }
    printf("Strings are equal!\n");
    return 1;
}

__global__ void find_longest_match_kernel(unsigned char *data, int i, uint16_t *longest_match_array, int len) {
    
    uint orig_i = i;
    uint cur = threadIdx.x + max(0, i - WINDOW_SIZE);
    uint16_t orig_cur = cur;
    // create the struct for storing the longest word we are currently look at
    uint length_of_match = 0;
    uint longest_match = 0;
    uint start = 0;
    while(cur < i && i < len && length_of_match < MATCH_LENGTH_MASK){
        // we have a match
        if(data[cur] == data[i]){
            cur++;
            i++;
            length_of_match++;
        }else{
            // we have a new longest match
            if (longest_match < length_of_match){
                longest_match = length_of_match;
                length_of_match = 0;
                start = cur;
            }
            break;
        }
    }
    // dont compress size 0 or 1
    if (longest_match < 2){
        longest_match_array[orig_cur] = 0;
    }
    // compress case
    else{
        uint16_t ret = 0;
        // bit shifting based on header of file
        ret = ((orig_i - ((start - longest_match))) << 4) | (longest_match);
        longest_match_array[orig_cur] = ret;
    }
}

// use a reduction function to find the max value in an array
// set the first element to this max
// we only want to base our max on size rather than distance
__global__ void find_max_reduction_kernel(uint16_t* data, int data_size){
    // we launch one thread per index
    int idx = threadIdx.x;
    if (idx < data_size){
        for(int stride = 1; stride < data_size; stride *= 2) {
            if (idx % (2*stride) == 0) {
                uint16_t lhs = get_length(data[idx]);
                uint16_t rhs = get_length(data[idx + stride]);
                data[idx] = lhs < rhs ? data[idx + stride] : data[idx];
            }
            __syncthreads();
        }
    }
}

uint16_t seq_find_longest_match(std::vector<unsigned char> data, int i){
    uint orig_i = i;
    uint cur = max(0, i - WINDOW_SIZE);
    // create the struct for storing the longest word we are currently look at
    uint length_of_match = 0;
    uint longest_match = 0;
    uint start = 0;
    while(cur < i && i < data.size()){
        if(data.at(i) == data.at(cur) && length_of_match <= MATCH_LENGTH_MASK){
            // increment i and cur since we have a match
            i++;
            cur++;
            length_of_match++;
        }else{
            // check if longest string so far
            if (longest_match < length_of_match){
                // new length
                start = cur;
                longest_match = length_of_match;
                // break early if we find max length match
                if (longest_match >= MATCH_LENGTH_MASK)
                    break;
            }
            // reset length of current match 
            length_of_match = 0;
            cur += 1;
            i = orig_i;
        }
    } 

    // 12 bytes = distance and 4 bytes is length
    if(longest_match < 2){
        return 0;
    }

    uint16_t ret = 0;
    // use our custom way of storing length and start of match in 2 bytes
    ret = ((orig_i - ((start - longest_match))) << 4) | (longest_match);

    return ret;
}


// keep an array of longest matches. 
// have each index select the length of the match  
uint16_t find_longest_match(unsigned char *d_data, int input_size, int i){
    int number_of_threads = WINDOW_SIZE + 1;

    uint16_t *device_longest_match_array;
    uint longest_match_array_length = min(WINDOW_SIZE, i + 1);
    // set things to 0 if not useful so no need to worry
    hipMalloc(&device_longest_match_array, 2 * longest_match_array_length);
    // invoke our kernel
    find_longest_match_kernel<<<1, min(i+1, number_of_threads)>>>(d_data, i, device_longest_match_array, input_size);
    // find max in using interleave
    find_max_reduction_kernel<<<1, min(i+1, number_of_threads)>>>(device_longest_match_array, input_size);
    // alloc host mem
    uint16_t res;
    hipMemcpy(&res, device_longest_match_array, 2, hipMemcpyDeviceToHost);

    // only need to copy one item
    // cudaMemcpy(host_longest_match_array, device_longest_match_array, 2*longest_match_array_length, cudaMemcpyDeviceToHost);

    // cudaFree(device_longest_match_array);

    return res;
}


// unsigned char is the representation of bytes that we will be using
std::vector<unsigned char> compress(unsigned char *d_data, std::vector<unsigned char> data, std::vector<bool> &compress_flags, int input_size){
    int i = 0;
    std::vector<unsigned char> output_buffer;
    // loop through all char in text
    while (i < input_size){
        uint16_t longest_match;
        // granularity tells us if we should run code in parallel or sequentially
        if (i > GRANULARITY){
            longest_match = seq_find_longest_match(data, i);
        }else{
            longest_match = find_longest_match(d_data, input_size, i);
        }
        uint16_t match_length = get_length(longest_match); 
        // item needs to be longer than 2 in order to save data
        if(match_length > 2){
            // 12 bytes = distance and 4 bytes is length
            uint16_t match_distance = get_distance(longest_match);
            // uint16_t match_length = get_length(longest_match); 
            // indicate that we are compressing
            compress_flags[i] = 1;
            output_buffer.push_back((longest_match >> 8) & 0xFF);
            output_buffer.push_back(longest_match & 0xFF);

            i += match_length;
        }else{
            output_buffer.push_back(data.at(i));
            i += 1;
        }
    }
    return output_buffer;
}

// decompress unsigned char vector streams 
// actual structure of data compression is described in header
std::vector<unsigned char> decompress(std::vector<unsigned char> data, std::vector<bool> &compress_flags){
    int compress_index = 0;
    uint data_index = 0;
    std::vector<unsigned char> output;

    while(data_index < data.size()){
        // item is compressed
        // printf("%d\n", compress_flags.at(compress_index));
        if(compress_flags.at(compress_index) == 1){
            uint16_t p1 = data[data_index];
            uint16_t p2 = data[data_index+1];
            uint16_t compressed_data = ((p1 << 8) & 0xFF00) | p2;
            uint16_t match_distance = get_distance(compressed_data);
            uint16_t match_length = get_length(compressed_data); 

            for(int i = 0; i < match_length; i++){
                output.push_back(output[((compress_index - match_distance) + i)]);
            }
            // two bytes for compressed
            data_index += 2;
            compress_index += match_length;
        }
        // item isn't compressed
        else{
            output.push_back(data[data_index]);
            // one byte for non compressed
            data_index++;
            compress_index++;
        }
    }
    return output;
}

int main(int argc, char *argv[]) {
    std::ifstream inFile;
    inFile.open(argv[1]); 
    
    std::stringstream strStream;
    auto read_file_start_time = std::chrono::high_resolution_clock::now();
    //read the file
    strStream << inFile.rdbuf(); 
    std::string s = strStream.str(); 
    // text input
    vector<unsigned char> input(s.begin(), s.end());

    // copy data into GPU
    uint input_size = ((int)input.size());
    unsigned char *d_data;
    hipMalloc(&d_data, input_size);
    hipMemcpy(d_data, input.data(), input_size, hipMemcpyHostToDevice);

    auto read_file_end_time = std::chrono::high_resolution_clock::now();

    // compression flags all init at 0
    vector<bool> compress_flags(input.size(), 0);
    
    auto compression_start_time = std::chrono::high_resolution_clock::now();
    auto compressed_buf = compress(d_data, input, compress_flags, input_size);
    auto compression_end_time = std::chrono::high_resolution_clock::now();
    hipFree(d_data);

    auto decompression_start_time = std::chrono::high_resolution_clock::now();
    auto output = decompress(compressed_buf, compress_flags);
    auto decompression_end_time = std::chrono::high_resolution_clock::now();

    // verify results
    bool res = are_vectors_equal(output, input);

    // save time durations
    auto compression_duration = (std::chrono::duration_cast<std::chrono::microseconds>(compression_end_time - compression_start_time).count()/pow(10,6));
    auto decompression_duration = (std::chrono::duration_cast<std::chrono::microseconds>(decompression_end_time - decompression_start_time).count()/pow(10,6));
    auto read_duration = (std::chrono::duration_cast<std::chrono::microseconds>(read_file_end_time - read_file_start_time).count()/pow(10,6));
    // write to output file

    ofstream myfile;
    myfile.open("output.txt");

    // report duartion inside outfile
    myfile << "Reading Input & Copying to GPU took: " << read_duration << " seconds" << std::endl;
    myfile << "Compression took: " << compression_duration << " seconds" << std::endl;
    myfile << "Decompression took: " << decompression_duration << " seconds" << std::endl;
    myfile.close();
    
    return 1;
}
